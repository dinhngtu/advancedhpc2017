#include "hip/hip_runtime.h"
#include <stdio.h>
#include <include/labwork.h>
#include <hip/hip_runtime_api.h>
#include <omp.h>

#define ACTIVE_THREADS 4

int main(int argc, char **argv) {
    //printf("USTH ICT Master 2017, Advanced Programming for HPC.\n");
    if (argc < 2) {
        printf("Usage: labwork <lwNum> <inputImage>\n");
        printf("   lwNum        labwork number\n");
        printf("   inputImage   the input file name, in JPEG format\n");
        return 0;
    }

    int lwNum = atoi(argv[1]);
    std::string inputFilename;

    // pre-initialize CUDA to avoid incorrect profiling
    //printf("Warming up...\n");
    char *temp;
    hipMalloc(&temp, 1024);

    Labwork labwork;
    if (lwNum != 2 ) {
        inputFilename = std::string(argv[2]);
        labwork.loadInputImage(inputFilename);
    }

    //printf("Starting labwork %d\n", lwNum);
    Timer timer;
    timer.start();
    switch (lwNum) {
        case 1:
            labwork.labwork1_CPU();
            labwork.saveOutputImage("labwork2-cpu-out.jpg");
            printf("labwork 1 CPU elapsed %.1fms\n", timer.getElapsedTimeInMilliSec());
            timer.start();
            labwork.labwork1_OpenMP();
            labwork.saveOutputImage("labwork2-openmp-out.jpg");
            printf("labwork 1 OpenMP elapsed %.1fms\n", timer.getElapsedTimeInMilliSec());
            break;
        case 2:
            labwork.labwork2_GPU();
            break;
        case 3:
            if (labwork.labwork3_GPU()) {
                labwork.saveOutputImage("labwork3-gpu-out.jpg");
                printf("labwork 3 elapsed %.1fms\n", timer.getElapsedTimeInMilliSec());
            }
            break;
        case 4:
            if (labwork.labwork4_GPU()) {
                labwork.saveOutputImage("labwork4-gpu-out.jpg");
                printf("labwork 4 elapsed %.1fms\n", timer.getElapsedTimeInMilliSec());
            }
            break;
        case 5:
            labwork.labwork5_GPU();
            labwork.saveOutputImage("labwork5-gpu-out.jpg");
            break;
        case 6:
            labwork.labwork6_GPU();
            labwork.saveOutputImage("labwork6-gpu-out.jpg");
            break;
        case 7:
            labwork.labwork7_GPU();
            labwork.saveOutputImage("labwork7-gpu-out.jpg");
            break;
        case 8:
            labwork.labwork8_GPU();
            labwork.saveOutputImage("labwork8-gpu-out.jpg");
            break;
        case 9:
            labwork.labwork9_GPU();
            labwork.saveOutputImage("labwork9-gpu-out.jpg");
            break;
        case 10:
            labwork.labwork10_GPU();
            labwork.saveOutputImage("labwork10-gpu-out.jpg");
            break;
    }
    //printf("labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
}

void Labwork::loadInputImage(std::string inputFileName) {
    inputImage = jpegLoader.load(inputFileName);
}

void Labwork::saveOutputImage(std::string outputFileName) {
    jpegLoader.save(outputFileName, outputImage, inputImage->width, inputImage->height, 90);
}

void Labwork::labwork1_CPU() {
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));
    for (int j = 0; j < 100; j++) {		// let's do it 100 times, otherwise it's too fast!
        for (int i = 0; i < pixelCount; i++) {
            outputImage[i * 3] = (char) (((int) inputImage->buffer[i * 3] + (int) inputImage->buffer[i * 3 + 1] +
                                          (int) inputImage->buffer[i * 3 + 2]) / 3);
            outputImage[i * 3 + 1] = outputImage[i * 3];
            outputImage[i * 3 + 2] = outputImage[i * 3];
        }
    }
}

void Labwork::labwork1_OpenMP() {
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));
    for (int j = 0; j < 100; j++) {		// let's do it 100 times, otherwise it's too fast!
#pragma omp parallel for schedule(dynamic)
        for (int i = 0; i < pixelCount; i++) {
            outputImage[i * 3] = (char) (((int) inputImage->buffer[i * 3] + (int) inputImage->buffer[i * 3 + 1] +
                                          (int) inputImage->buffer[i * 3 + 2]) / 3);
            outputImage[i * 3 + 1] = outputImage[i * 3];
            outputImage[i * 3 + 2] = outputImage[i * 3];
        }
    }
}

int getSPcores(hipDeviceProp_t devProp) {
    int cores = 0;
    int mp = devProp.multiProcessorCount;
    switch (devProp.major) {
        case 2: // Fermi
            if (devProp.minor == 1) cores = mp * 48;
            else cores = mp * 32;
            break;
        case 3: // Kepler
            cores = mp * 192;
            break;
        case 5: // Maxwell
            cores = mp * 128;
            break;
        case 6: // Pascal
            if (devProp.minor == 1) cores = mp * 128;
            else if (devProp.minor == 0) cores = mp * 64;
            else printf("Unknown device type\n");
            break;
        default:
            printf("Unknown device type\n");
            break;
    }
    return cores;
}

void Labwork::labwork2_GPU() {
    int numDevices;
    if (hipGetDeviceCount(&numDevices) != hipSuccess) {
        fprintf(stderr, "cannot get number of devices\n");
        return;
    }
    printf("%d devices found\n", numDevices);
    for (int i = 0; i < numDevices; i++) {
        hipDeviceProp_t prop;
        if (hipGetDeviceProperties(&prop, i) != hipSuccess) {
            fprintf(stderr, "cannot get device props\n");
            return;
        }
        printf("Information for device %d:\n", i);
        printf("Device name: %s\n", prop.name);
        int cores = getSPcores(prop);
        printf("Core count: %d\n", cores);
        printf("Core clock rate: %d kHz\n", prop.clockRate);
        printf("Multiprocessor count: %d\n", prop.multiProcessorCount);
        printf("Warp size: %d threads\n", prop.warpSize);
        printf("Memory clock rate: %d kHz\n", prop.memoryClockRate);
        printf("Memory bus width: %d bits\n", prop.memoryBusWidth);
        printf("\n");
    }
}

__global__ void labwork3(uchar3 * __restrict__ input, uchar3 * __restrict__ output, long long pixelCount) {
    long long i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < pixelCount) {
        output[i].x = (char)(((int)input[i].x + input[i].y + input[i].z) / 3);
        output[i].y = output[i].z = output[i].x;
    }
}

int Labwork::labwork3_GPU() {
    long long pixelCount = inputImage->width * inputImage->height;
    char *blockSizeEnv = getenv("LW3_CUDA_BLOCK_SIZE");
    if (!blockSizeEnv) {
        fprintf(stderr, "invalid block size\n");
        return 0;
    }
    int blockSize = atoi(blockSizeEnv);
    long long numBlocks = pixelCount / blockSize + 1;

    uchar3 *inputCudaBuffer;
    if (hipMalloc(&inputCudaBuffer, pixelCount * sizeof(uchar3)) != hipSuccess) {
        fprintf(stderr, "memory allocation error\n");
        return 0;
    }
    uchar3 *outputCudaBuffer;
    if (hipMalloc(&outputCudaBuffer, pixelCount * sizeof(uchar3)) != hipSuccess) {
        fprintf(stderr, "memory allocation error\n");
        return 0;
    }

    outputImage = static_cast<char *>(malloc(pixelCount * 3));
    if (hipMemcpy(inputCudaBuffer, inputImage->buffer, pixelCount * sizeof(uchar3), hipMemcpyHostToDevice) != hipSuccess) {
        fprintf(stderr, "input buffer copy error\n");
        return 0;
    }
    for (int j = 0; j < 100; j++) {
        labwork3<<<numBlocks, blockSize>>>(inputCudaBuffer, outputCudaBuffer, pixelCount);
    }
    hipDeviceSynchronize();
    if (hipMemcpy(outputImage, outputCudaBuffer, pixelCount * sizeof(uchar3), hipMemcpyDeviceToHost) != hipSuccess) {
        fprintf(stderr, "output buffer copy error\n");
        return 0;
    }

    hipFree(inputCudaBuffer);
    hipFree(outputCudaBuffer);

    return 1;
}

__global__ void labwork4(uchar3 * __restrict__ input, uchar3 * __restrict__ output, long long pixelCount, int width) {
    long row = blockIdx.y * gridDim.y + threadIdx.y;
    long long i = row * width + threadIdx.x;
    if (i < pixelCount) {
        output[i].x = (char)(((int)input[i].x + input[i].y + input[i].z) / 3);
        output[i].y = output[i].z = output[i].x;
    }
}

int Labwork::labwork4_GPU() {
    long long pixelCount = inputImage->width * inputImage->height;
    char *blockSizeEnv = getenv("LW4_CUDA_BLOCK_SIZE");
    if (!blockSizeEnv) {
        fprintf(stderr, "invalid block size\n");
        return 0;
    }
    int blockSize = atoi(blockSizeEnv);

    long gridWidth = (inputImage->width + blockSize - 1) / blockSize;
    long gridHeight = (inputImage->width + blockSize - 1) / blockSize;
    dim3 gdim(gridWidth, gridHeight);
    dim3 bdim(blockSize, blockSize);

    uchar3 *inputCudaBuffer;
    if (hipMalloc(&inputCudaBuffer, pixelCount * sizeof(uchar3)) != hipSuccess) {
        fprintf(stderr, "memory allocation error\n");
        return 0;
    }
    uchar3 *outputCudaBuffer;
    if (hipMalloc(&outputCudaBuffer, pixelCount * sizeof(uchar3)) != hipSuccess) {
        fprintf(stderr, "memory allocation error\n");
        return 0;
    }

    outputImage = static_cast<char *>(malloc(pixelCount * 3));
    if (hipMemcpy(inputCudaBuffer, inputImage->buffer, pixelCount * sizeof(uchar3), hipMemcpyHostToDevice) != hipSuccess) {
        fprintf(stderr, "input buffer copy error\n");
        return 0;
    }
    for (int j = 0; j < 100; j++) {
        labwork4<<<gdim, bdim>>>(inputCudaBuffer, outputCudaBuffer, pixelCount, inputImage->width);
    }
    hipDeviceSynchronize();
    if (hipMemcpy(outputImage, outputCudaBuffer, pixelCount * sizeof(uchar3), hipMemcpyDeviceToHost) != hipSuccess) {
        fprintf(stderr, "output buffer copy error\n");
        return 0;
    }

    hipFree(inputCudaBuffer);
    hipFree(outputCudaBuffer);

    return 1;
}

void Labwork::labwork5_GPU() {
    
}

void Labwork::labwork6_GPU() {

}

void Labwork::labwork7_GPU() {

}

void Labwork::labwork8_GPU() {

}

void Labwork::labwork9_GPU() {

}

void Labwork::labwork10_GPU() {

}
